#include <iostream>

#include "gpma.cuh"
#include "gpma_bfs.cuh"

void load_data(const char *file_path, thrust::host_vector<int> &host_x, thrust::host_vector<int> &host_y,
        int &node_size, int &edge_size) {

    FILE *fp;
    fp = fopen(file_path, "r");
    if (not fp) {
        printf("Open graph file failed.\n");
        exit(0);
    }

    fscanf(fp, "%d %d", &node_size, &edge_size);
    printf("node_num: %d, edge_num: %d\n", node_size, edge_size);

    host_x.resize(edge_size);
    host_y.resize(edge_size);

    for (int i = 0; i < edge_size; i++) {
        int x, y;
        (void) fscanf(fp, "%d %d", &x, &y);
        host_x[i] = x;
        host_y[i] = y;
    }

    printf("Graph file is loaded.\n");
    fclose(fp);
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("Invalid arguments.\n");
        return -1;
    }

    char* data_path = argv[1];
    int bfs_start_node = std::atoi(argv[2]);

    hipDeviceSetLimit(hipLimitMallocHeapSize, 1024ll * 1024 * 1024);
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 5);

    thrust::host_vector<int> host_x;
    thrust::host_vector<int> host_y;
    int node_size;
    int edge_size;
    load_data(data_path, host_x, host_y, node_size, edge_size);

    int half = edge_size / 2;
    thrust::host_vector<KEY_TYPE> h_base_keys(half);
    for (int i = 0; i < half; i++) {
        h_base_keys[i] = ((KEY_TYPE) host_x[i] << 32) + host_y[i];
    }

    NATIVE_VEC_KEY<GPU> base_keys = h_base_keys;
    NATIVE_VEC_VALUE<GPU> base_values(half, 1);
    hipDeviceSynchronize();

    int num_slide = 100;
    int step = half / num_slide;

    LOG_TIME("before init_csr_gpma")
    GPMA<GPU> gpma(node_size);
    hipDeviceSynchronize();

    LOG_TIME("before update_gpma 1")
    update_gpma(gpma, base_keys, base_values);
    thrust::device_vector<SIZE_TYPE> bfs_result(node_size);
    hipDeviceSynchronize();

    LOG_TIME("before first bfs")
    gpma_bfs(RAW_PTR(gpma.keys), RAW_PTR(gpma.values), RAW_PTR(gpma.row_offset), node_size,
            edge_size, bfs_start_node, RAW_PTR(bfs_result));
    int reach_nodes = node_size - thrust::count(bfs_result.begin(), bfs_result.end(), 0);
    printf("start from node %d, number of reachable nodes: %d\n", bfs_start_node, reach_nodes);
    LOG_TIME_2("===============BEGIN MAIN LOOP==================")

    LOG_TIME("before main loop")
    for (int i = 0; i < num_slide; i++) {
        thrust::host_vector<KEY_TYPE> hk(step * 2);
        for (int j = 0; j < step; j++) {
            int idx = half + i * step + j;
            hk[j] = ((KEY_TYPE) host_x[idx] << 32) + host_y[idx];
        }
        for (int j = 0; j < step; j++) {
            int idx = i * step + j;
            hk[j + step] = ((KEY_TYPE) host_x[idx] << 32) + host_y[idx];
        }

        NATIVE_VEC_VALUE<GPU> update_values(step * 2);
        thrust::fill(update_values.begin(), update_values.begin() + step, 1);
        thrust::fill(update_values.begin() + step, update_values.end(), VALUE_NONE);
        NATIVE_VEC_KEY<GPU> update_keys = hk;
        hipDeviceSynchronize();

        update_gpma(gpma, update_keys, update_values);
        hipDeviceSynchronize();
    }
    LOG_TIME_2("===============END MAIN LOOP==================")
    printf("Graph is updated.\n");
    LOG_TIME("before second bfs")

    gpma_bfs(RAW_PTR(gpma.keys), RAW_PTR(gpma.values), RAW_PTR(gpma.row_offset), node_size,
            edge_size, bfs_start_node, RAW_PTR(bfs_result));
    reach_nodes = node_size - thrust::count(bfs_result.begin(), bfs_result.end(), 0);
    printf("start from node %d, number of reachable nodes: %d\n", bfs_start_node, reach_nodes);
    LOG_TIME("after second loop")

    return 0;
}
