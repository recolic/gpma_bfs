#include <iostream>

#include "gpma.cuh"
#include "gpma_bfs.cuh"

void load_data(const char *file_path, thrust::host_vector<int> &host_x, thrust::host_vector<int> &host_y, int &node_size, int &edge_size) {

    FILE *fp;
    fp = fopen(file_path, "r");
    if (not fp) {
        printf("Open graph file failed.\n");
        exit(0);
    }

    fscanf(fp, "%d %d", &node_size, &edge_size);
    printf("node_num: %d, edge_num: %d\n", node_size, edge_size);

    host_x.resize(edge_size);
    host_y.resize(edge_size);

    for (int i = 0; i < edge_size; i++) {
        int x, y;
        (void)fscanf(fp, "%d %d", &x, &y);
        host_x[i] = x;
        host_y[i] = y;
    }

    printf("Graph file is loaded.\n");
    fclose(fp);
}

int main(int argc, char **argv) {
    if (argc != 3) {
        printf("Invalid arguments.\n");
        return -1;
    }

    char *data_path = argv[1];
    int bfs_start_node = std::atoi(argv[2]);

    hipDeviceSetLimit(hipLimitMallocHeapSize, 1024ll * 1024 * 700);
    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 5);

    thrust::host_vector<int> host_x;
    thrust::host_vector<int> host_y;
    int node_size;
    int edge_size;
    load_data(data_path, host_x, host_y, node_size, edge_size);

    int half = edge_size / 2;
    thrust::host_vector<KEY_TYPE> h_base_keys(half);
    for (int i = 0; i < half; i++) {
        h_base_keys[i] = ((KEY_TYPE)host_x[i] << 32) + host_y[i];
    }

    NATIVE_VEC_KEY<CPU> base_keys = h_base_keys;
    NATIVE_VEC_VALUE<CPU> base_values(half, 1);
    hipDeviceSynchronize();

    int num_slide = 100;
    int step = half / num_slide;

    LOG_TIME("before init_csr_gpma")
    constexpr size_t cpu_count = 4;
#ifndef TEST_CPUS
#define TEST_CPUS cpu_count-1
#define TEST_GPUS 1
#endif
    GPMA_multidev<TEST_CPUS, TEST_GPUS> gpma(node_size);
    hipDeviceSynchronize();

    LOG_TIME("before update_gpma 1")
    gpma.update_batch(base_keys, base_values);
    native_vector<CPU, SIZE_TYPE> bfs_result(node_size);
    hipDeviceSynchronize();

    LOG_TIME("before first bfs")
#ifndef _DISABLE_BFS
    gpma_bfs(gpma, node_size, edge_size, bfs_start_node, RAW_PTR(bfs_result));
    int reach_nodes = node_size - thrust::count(bfs_result.begin(), bfs_result.end(), 0);
    printf("start from node %d, number of reachable nodes: %d\n", bfs_start_node, reach_nodes);
#endif

    LOG_TIME("before main loop")
    for (int i = 0; i < num_slide; i++) {
        thrust::host_vector<KEY_TYPE> hk(step * 2);
        for (int j = 0; j < step; j++) {
            int idx = half + i * step + j;
            hk[j] = ((KEY_TYPE)host_x[idx] << 32) + host_y[idx];
        }
        for (int j = 0; j < step; j++) {
            int idx = i * step + j;
            hk[j + step] = ((KEY_TYPE)host_x[idx] << 32) + host_y[idx];
        }

        NATIVE_VEC_VALUE<CPU> update_values(step * 2);
        thrust::fill(update_values.begin(), update_values.begin() + step, 1);
        thrust::fill(update_values.begin() + step, update_values.end(), VALUE_NONE);
        NATIVE_VEC_KEY<CPU> update_keys = hk;
        hipDeviceSynchronize();

        gpma.update_batch(update_keys, update_values);
        hipDeviceSynchronize();
    }
    printf("Graph is updated.\n");
    LOG_TIME("before second bfs")

#ifndef _DISABLE_BFS
    gpma_bfs(gpma, node_size, edge_size, bfs_start_node, RAW_PTR(bfs_result));
    reach_nodes = node_size - thrust::count(bfs_result.begin(), bfs_result.end(), 0);
    printf("start from node %d, number of reachable nodes: %d\n", bfs_start_node, reach_nodes);
#endif
    LOG_TIME("after second bfs")

    return 0;
}
